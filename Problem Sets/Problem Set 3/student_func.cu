#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.


  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"
#include <limits>

template <typename reduce_op>
__global__ void reduce_kernel(const float *const d_logLuminance,
                              float *d_output, const size_t numElems,
                              reduce_op op) {
  int gid = threadIdx.x + blockIdx.x * blockDim.x;
  int tid = threadIdx.x;

  extern __shared__ float s_data[];

  if (gid < numElems) {
    s_data[tid] = d_logLuminance[gid];
  }
  __syncthreads();

  int offset = blockIdx.x * blockDim.x;
  int nsize =
      (offset + blockDim.x < numElems) ? blockDim.x : (numElems - offset);
  while (nsize > 1) {
    int len = (nsize + 1) / 2;
    if (tid < nsize / 2) {
      s_data[tid] = op(s_data[tid], s_data[tid + len]);
    }
    nsize = len;
    __syncthreads();
  }

  if (tid == 0) {
    d_output[blockIdx.x] = s_data[0];
  }
}

template <typename reduce_op>
float reduce(const float *const d_logLuminance, int numElems, float init,
             const reduce_op &op) {
  int block_size = 1024;
  int grid_size = (numElems + block_size - 1) / block_size;

  float *d_output;
  checkCudaErrors(hipMalloc(&d_output, grid_size * sizeof(float)));

  reduce_kernel<<<grid_size, block_size, block_size * sizeof(float)>>>(
      d_logLuminance, d_output, numElems, op);
  hipDeviceSynchronize();
  checkCudaErrors(hipGetLastError());
  printf("first round, grid_size: %d\n", grid_size);

  numElems = grid_size;
  for (; numElems >= block_size; numElems = grid_size) {
    grid_size = (numElems + block_size - 1) / block_size;
    printf("grid_size: %d\n", grid_size);

    float *d_input = d_output;
    checkCudaErrors(hipMalloc(&d_output, grid_size * sizeof(float)));

    reduce_kernel<<<grid_size, block_size, block_size * sizeof(float)>>>(
        d_input, d_output, numElems, op);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipFree(d_input));
  }

  printf("grid_size: %d, numElems: %d\n", grid_size, numElems);
  float *h_output = new float[numElems];
  checkCudaErrors(hipMemcpy(h_output, d_output, numElems * sizeof(float),
                             hipMemcpyDeviceToHost));
  hipFree(d_output);

  float res = init;
  for (int i = 0; i < numElems; ++i) {
    res = op(res, h_output[i]);
  }
  delete[] h_output;

  return res;
}

__global__ void histogram(const float *const d_logLuminance, size_t numElems,
                          unsigned int *const d_cdf, size_t numBins,
                          float min_logLum, float max_logLum) {
  int gid = threadIdx.x + blockIdx.x * blockDim.x;
  if (gid < numElems) {
    int bin = (d_logLuminance[gid] - min_logLum) / (max_logLum - min_logLum) *
              numBins;
    atomicAdd(&d_cdf[bin], 1);
  }
}

__global__ void HS_scan_kernel(unsigned int *d_cdf, unsigned int *d_out,
                               int numBins, int stride) {
  int gid = threadIdx.x + blockIdx.x * blockDim.x;
  if (gid < numBins) {
    if (gid - stride < 0) {
      d_out[gid] = d_cdf[gid];
    } else {
      d_out[gid] = d_cdf[gid] + d_cdf[gid - stride];
    }
  }
}

// Hillis & Steele inclusive scan
void HS_scan_impl(unsigned int **d_cdf, unsigned int **d_out, int numBins,
                  int block_size) {
  for (int stride = 1; stride < numBins; stride <<= 1) {
    // swith input and output every step
    if (stride != 1) {
      unsigned int *temp = *d_cdf;
      *d_cdf = *d_out;
      *d_out = temp;
    }

    int grid_size = (numBins + block_size - 1) / block_size;
    HS_scan_kernel<<<grid_size, block_size>>>(*d_cdf, *d_out, numBins, stride);

    // TODO: should just copy d_cdf to d_out to reduce threads num in next step?
  }
}

void HS_scan(unsigned int *const h_cdf, int numBins, int block_size) {
  // need an out array to avoid data race
  unsigned int *d_cdf, *d_out;
  checkCudaErrors(hipMalloc(&d_cdf, numBins * sizeof(unsigned int)));
  checkCudaErrors(hipMalloc(&d_out, numBins * sizeof(unsigned int)));

  checkCudaErrors(hipMemcpy(d_cdf, h_cdf, numBins * sizeof(unsigned int),
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_out, h_cdf, numBins * sizeof(unsigned int),
                             hipMemcpyHostToDevice));

  HS_scan_impl(&d_cdf, &d_out, numBins, block_size);
  checkCudaErrors(hipMemcpy(h_cdf, d_out, numBins * sizeof(unsigned int),
                             hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(d_cdf));
  checkCudaErrors(hipFree(d_out));
}

__global__ void UpSweep(unsigned int *const d_cdf, unsigned int *const d_sums) {
  extern __shared__ unsigned int temp[];

  int tid = threadIdx.x;
  int gid = threadIdx.x + 2 * blockIdx.x * blockDim.x;
  temp[tid] = d_cdf[gid];
  temp[tid + blockDim.x] = d_cdf[gid + blockDim.x];
  __syncthreads();

  for (int stride = 1; stride < 2 * blockDim.x; stride <<= 1) {
    int idx = (tid + 1) * stride * 2 - 1;
    if (idx < 2 * blockDim.x) {
      temp[idx] = temp[idx - stride] + temp[idx];
    }

    __syncthreads();
  }

  d_cdf[gid] = temp[tid];
  d_cdf[gid + blockDim.x] = temp[tid + blockDim.x];

  if (tid == 0 && d_sums != NULL) {
    d_sums[blockIdx.x] = temp[2 * blockDim.x - 1];
  }
}

__global__ void DownSweep(unsigned int *const d_cdf) {
  extern __shared__ unsigned int temp[];

  int tid = threadIdx.x;
  int gid = threadIdx.x + 2 * blockIdx.x * blockDim.x;
  temp[tid] = d_cdf[gid];
  temp[tid + blockDim.x] = d_cdf[gid + blockDim.x];
  __syncthreads();

  temp[2 * blockDim.x - 1] = 0;

  for (int stride = blockDim.x; stride >= 1; stride >>= 1) {
    int idx = (tid + 1) * stride * 2 - 1;
    if (idx < 2 * blockDim.x) {
      int t = temp[idx - stride];
      temp[idx - stride] = temp[idx];
      temp[idx] += t;
    }

    __syncthreads();
  }

  d_cdf[gid] = temp[tid];
  d_cdf[gid + blockDim.x] = temp[tid + blockDim.x];
}

__global__ void BlockSum(unsigned int *const d_cdf,
                         unsigned int *const d_sums) {
  int gid = threadIdx.x + 2 * blockIdx.x * blockDim.x;
  d_cdf[gid] += d_sums[blockIdx.x];
  d_cdf[gid + blockDim.x] += d_sums[blockIdx.x];
}

// REQURES: d_cdf padding to 2 * block_size
void Blelloch_scan_impl(unsigned int *const d_cdf, int numBins,
                        int block_size) {
  int grid_size = numBins / (2 * block_size);
  int share_size = 2 * block_size * sizeof(unsigned int);

  if (grid_size == 1) {
    UpSweep<<<grid_size, block_size, share_size>>>(d_cdf, NULL);
    DownSweep<<<grid_size, block_size, share_size>>>(d_cdf);
  } else {
    unsigned int *d_sums;
    // padding block sums for futher Blelloch scan
    int grid_size_ = (grid_size + 2 * block_size - 1) / (2 * block_size);
    int padding = 2 * grid_size_ * block_size - grid_size;
    checkCudaErrors(
        hipMalloc(&d_sums, (grid_size + padding) * sizeof(unsigned int)));

    if (padding > 0) {
      checkCudaErrors(
          hipMemset(d_sums + grid_size, 0, padding * sizeof(unsigned int)));
    }

    UpSweep<<<grid_size, block_size, share_size>>>(d_cdf, d_sums);
    DownSweep<<<grid_size, block_size, share_size>>>(d_cdf);

    Blelloch_scan_impl(d_sums, grid_size + padding, block_size);
    BlockSum<<<grid_size, block_size>>>(d_cdf, d_sums);

    checkCudaErrors(hipFree(d_sums));
  }
}

// Blelloch exclusive scan
// reference: https://developer.nvidia.com/gpugems/GPUGems3/gpugems3_ch39.html
void Blelloch_scan(unsigned int *const h_cdf, int numBins, int block_size) {
  unsigned int *d_cdf;
  // each thread block will handle two data block
  int grid_size = (numBins + 2 * block_size - 1) / (2 * block_size);
  int padding = 2 * grid_size * block_size - numBins;

  checkCudaErrors(
      hipMalloc(&d_cdf, (numBins + padding) * sizeof(unsigned int)));
  checkCudaErrors(hipMemcpy(d_cdf, h_cdf, numBins * sizeof(unsigned int),
                             hipMemcpyHostToDevice));

  // padding last block
  if (padding > 0) {
    checkCudaErrors(
        hipMemset(d_cdf + numBins, 0, padding * sizeof(unsigned int)));
  }

  Blelloch_scan_impl(d_cdf, numBins + padding, block_size);

  checkCudaErrors(hipMemcpy(h_cdf, d_cdf, numBins * sizeof(unsigned int),
                             hipMemcpyDeviceToHost));
  checkCudaErrors(hipFree(d_cdf));
}

void your_histogram_and_prefixsum(const float *const d_logLuminance,
                                  unsigned int *const d_cdf, float &min_logLum,
                                  float &max_logLum, const size_t numRows,
                                  const size_t numCols, const size_t numBins) {
  // TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */
  min_logLum = reduce(
      d_logLuminance, numRows * numCols, std::numeric_limits<float>::max(),
      [] __host__ __device__(float a, float b) { return (a < b) ? a : b; });

  printf("minimum: %f\n", min_logLum);

  max_logLum = reduce(
      d_logLuminance, numRows * numCols, std::numeric_limits<float>::min(),
      [] __host__ __device__(float a, float b) { return (a < b) ? b : a; });

  printf("maximum: %f\n", max_logLum);

  int numElems = numRows * numCols;
  int block_size = 1024;
  int grid_size = (numElems + block_size - 1) / block_size;

  histogram<<<grid_size, block_size>>>(d_logLuminance, numElems, d_cdf, numBins,
                                       min_logLum, max_logLum);

  Blelloch_scan_impl(d_cdf, numBins, block_size);
}